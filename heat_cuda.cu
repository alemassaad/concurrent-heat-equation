#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <time.h>
#include <assert.h>
#include "initialize.h"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

__device__ int getIndex(const int i, const int j, const int width) {
    return i * width + j;
}

__global__ void update(float* U, float* U_next, int nx, int ny, float lambda) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int i = blockIdx.x * blockDim.x + tx;
    int j = blockIdx.y * blockDim.y + ty;

    // Shared memory allocation with halo cells
    __shared__ float s_U[BLOCK_SIZE_X + 2][BLOCK_SIZE_Y + 2];

    // Load data into shared memory, including halo cells
    if (i < nx && j < ny) {
        s_U[tx + 1][ty + 1] = U[getIndex(i, j, ny)];

        if (tx == 0 && i > 0) s_U[0][ty + 1] = U[getIndex(i - 1, j, ny)];
        if (tx == BLOCK_SIZE_X - 1 && i < nx - 1) s_U[tx + 2][ty + 1] = U[getIndex(i + 1, j, ny)];
        if (ty == 0 && j > 0) s_U[tx + 1][0] = U[getIndex(i, j - 1, ny)];
        if (ty == BLOCK_SIZE_Y - 1 && j < ny - 1) s_U[tx + 1][ty + 2] = U[getIndex(i, j + 1, ny)];

        if (tx == 0 && ty == 0 && i > 0 && j > 0) s_U[0][0] = U[getIndex(i - 1, j - 1, ny)];
        if (tx == 0 && ty == BLOCK_SIZE_Y - 1 && i > 0 && j < ny - 1) s_U[0][ty + 2] = U[getIndex(i - 1, j + 1, ny)];
        if (tx == BLOCK_SIZE_X - 1 && ty == 0 && i < nx - 1 && j > 0) s_U[tx + 2][0] = U[getIndex(i + 1, j - 1, ny)];
        if (tx == BLOCK_SIZE_X - 1 && ty == BLOCK_SIZE_Y - 1 && i < nx - 1 && j < ny - 1) s_U[tx + 2][ty + 2] = U[getIndex(i + 1, j + 1, ny)];

        __syncthreads();

        if (i > 0 && i < nx - 1 && j > 0 && j < ny - 1) {
            float uij = s_U[tx + 1][ty + 1];
            float uim1j = s_U[tx][ty + 1];
            float uip1j = s_U[tx + 2][ty + 1];
            float uijm1 = s_U[tx + 1][ty];
            float uijp1 = s_U[tx + 1][ty + 2];

            float term = (1 - 4 * lambda) * uij + lambda * (uim1j + uip1j + uijm1 + uijp1);
            U_next[getIndex(i, j, ny)] = isnan(term) ? uij : term;
        }
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(1);
    }
}

void test_filename_truncation() {
    char filename[256];
    int needed = snprintf(filename, sizeof(filename), "output_seq/very_long_filename_prefix_%d.dat", INT_MAX);
    assert(needed < sizeof(filename));
    printf("Test passed: Filename is within buffer limits.\n");
}

int main(int argc, char **argv) {
    if (argc < 2) {
        fprintf(stderr, "Usage: %s <run_num>\n", argv[0]);
        return 1;
    }

    int run_num = atoi(argv[1]);

    int size = NX * NY * sizeof(float);
    float* U = (float*)calloc(NX * NY, sizeof(float));
    float* U_next = (float*)calloc(NX * NY, sizeof(float));
    float* d_U, * d_U_next;

    checkCudaError(hipMalloc((void**)&d_U, size), "Failed to allocate device memory for U");
    checkCudaError(hipMalloc((void**)&d_U_next, size), "Failed to allocate device memory for U_next");

    initialize(U, NX, NY);

    for (int i = 0; i < NX * NY; i++) {
        if (isnan(U[i])) {
            printf("Initialization produced NaN at index %d\n", i);
            free(U);
            free(U_next);
            return 1;
        }
    }

    checkCudaError(hipMemcpy(d_U, U, size, hipMemcpyHostToDevice), "Failed to copy U to device");
    checkCudaError(hipMemcpy(d_U_next, U, size, hipMemcpyHostToDevice), "Failed to copy U_next to device");

    float lambda = GAMMA / (DELTA * DELTA);
    if (lambda >= 0.5) {
        printf("Warning: lambda = %f is approaching instability, adjusting...\n", lambda);
        lambda = 0.49; // Adjust lambda to maintain stability
    }
    
    dim3 threadsPerBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 numBlocks((NX + threadsPerBlock.x - 1) / threadsPerBlock.x, (NY + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Create output directory if it does not exist
    struct stat st = {0};
    if (stat("output_cuda", &st) == -1) {
        mkdir("output_cuda", 0700);
        printf("Created directory: output_cuda\n");
    }
    if (stat("frames_cuda", &st) == -1) {
        mkdir("frames_cuda", 0700);
        printf("Created directory: frames_cuda\n");
    }

    char csv_filename[256];
    snprintf(csv_filename, sizeof(csv_filename), "results/results_%dx%d.csv", NX, NY);

    int last_complete_step = -1;
    FILE *csv_file = fopen(csv_filename, "r");
    if (csv_file != NULL) {
        char line[256];
        while (fgets(line, sizeof(line), csv_file)) {
            int step;
            char *ptr = strrchr(line, ',');
            if (ptr && *(ptr + 1) == '\n') {
                sscanf(line, "%*d,%*[^,],%*d,%d,%*f", &step);
                last_complete_step = step;
            }
        }
        fclose(csv_file);
    }

    csv_file = fopen(csv_filename, "a");
    if (csv_file == NULL) {
        fprintf(stderr, "Error opening CSV file %s\n", csv_filename);
        return 1;
    }

    struct timespec start, end;
    double cumulative_time = 0.0;

    for (int step = last_complete_step + 1; step <= N_STEPS; step++) {
        clock_gettime(CLOCK_MONOTONIC, &start);
        update<<<numBlocks, threadsPerBlock>>>(d_U, d_U_next, NX, NY, lambda);
        checkCudaError(hipGetLastError(), "Kernel launch failed");
        checkCudaError(hipDeviceSynchronize(), "Kernel execution failed");
        clock_gettime(CLOCK_MONOTONIC, &end);

        // Swap pointers
        float* temp = d_U;
        d_U = d_U_next;
        d_U_next = temp;

        double elapsed_time;
        if (end.tv_nsec < start.tv_nsec) {
            elapsed_time = (end.tv_sec - start.tv_sec - 1) + (end.tv_nsec + 1e9 - start.tv_nsec) * 1e-9;
        } else {
            elapsed_time = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) * 1e-9;
        }

        cumulative_time += elapsed_time;

        if (step % STEP_INTERVAL == 0) {
            checkCudaError(hipMemcpy(U, d_U, size, hipMemcpyDeviceToHost), "Failed to copy U from device to host");

            char filename[100];
            sprintf(filename, "output_cuda/output_%d.dat", step);
            FILE* fp = fopen(filename, "w");
            if (fp == NULL) {
                printf("Error opening file %s\n", filename);
                exit(1);
            }
            for (int i = 0; i < NX; i++) {
                for (int j = 0; j < NY; j++) {
                    fprintf(fp, "%f ", U[i * NY + j]);
                }
                fprintf(fp, "\n");
            }
            fclose(fp);

            fprintf(csv_file, "%d,cuda,%d,%d,%f\n", NX, run_num, step, cumulative_time);
            printf("Done step: %d in %f seconds (cumulative time: %f)\n", step, elapsed_time, cumulative_time);
        }
    }

    hipFree(d_U);
    hipFree(d_U_next);
    free(U);
    free(U_next);

    fclose(csv_file);

    return 0;
}
